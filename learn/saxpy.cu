#include "hip/hip_runtime.h"
__global__
void saxpy(int n, float a, float * restrict x, float * restrict y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

...
int N = 1<<20;
hipMemcpy(d_x, x, N, hipMemcpyHostToDevice);
hipMemcpy(d_y, y, N, hipMemcpyHostToDevice);

// Perform SAXPY on 1M elements
saxpy<<<4096,256>>>(N, 2.0, d_x, d_y);

hipMemcpy(y, d_y, N, hipMemcpyDeviceToHost);
